#include "hip/hip_runtime.h"
/* ==================================================================
	Programmers:
	Kevin Wagner
	Elijah Malaby
	John Casey

	Omptimizing SDH histograms for input larger then global memory
   ==================================================================
*/

#include <stdio.h>
#include <stdlib.h>
#include <ctype.h>
#include <math.h>
#include <sys/time.h>


#define BOX_SIZE 23000 /* size of the data box on one dimension */
#define CHUNK_SIZE 1024 /*2^14* size of a single chunk of atoms/

/* descriptors for single atom in the tree */
typedef struct atomdesc {
	float x_pos;
	float y_pos;
	float z_pos;
} atom;

unsigned long long * histogram;		/* list of all buckets in the histogram */
unsigned long long  PDH_acnt;	/* total number of data points */
int block_size;		/* Number of threads per block */
int num_buckets;	/* total number of buckets in the histogram */
float   PDH_res;	/* value of w */
atom * atom_list;	/* list of all data points */
unsigned long long * histogram_GPU;
unsigned long long * temp_interchunk_histogram_GPU;
unsigned long long * temp_intrachunk_histogram_GPU;
atom * chunk_a;
atom * chunk_b;

/*Adds the InputHists to the hist*/
__global__ void kernelSumHistogram( unsigned long long int *InputHists, unsigned long long int *hist, int num_atoms, int num_buckets, int block_size) {
  unsigned long long int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int h_pos = tid;
  unsigned long long int NumberOfSumLoop = 0;
  NumberOfSumLoop = (num_atoms)/block_size + ((num_atoms%block_size) ? 1:0);

  while(h_pos < num_buckets) {
    unsigned long long int tmpAns = 0;
    for(int i=0;i<NumberOfSumLoop;i++){
      tmpAns = tmpAns + *(InputHists+(i*num_buckets)+h_pos);
    }
    hist[h_pos] = tmpAns;
    h_pos += blockDim.x * gridDim.x;
  }
  __syncthreads();
}

/*Compares all the atoms in block a with all the atoms in block b*/
__device__ void block_to_block (atom * block_a, int a_length, atom * block_b, int b_length, unsigned long long * histogram, float resolution) {
  atom me = block_a[threadIdx.x]; // Cache my atom
  for(int i = 0; i < b_length; i++) // Loop through the atoms in block b
    atomicAdd(&(histogram[(int)(sqrt((me.x_pos - block_b[i].x_pos) * (me.x_pos - block_b[i].x_pos) +
                                     (me.y_pos - block_b[i].y_pos) * (me.y_pos - block_b[i].y_pos) +
                                     (me.z_pos - block_b[i].z_pos) * (me.z_pos - block_b[i].z_pos)) / resolution)]),
              1); // Add to the histogram bucket for this pair (me, block_b[i])
}

__global__ void GPUInterChunkKernel (unsigned long long chunk_a_size, unsigned long long chunk_b_size, float histogram_resolution, atom * chunk_a, atom * chunk_b, unsigned long long * histogram_GPU, int num_buckets) {
  extern __shared__ unsigned long long SHist[];
  int i;
  atom * my_block = &chunk_a[blockIdx.x * blockDim.x]; // Pointer to this block's atoms in chunk a

  for(h_pos=threadIdx.x; h_pos < num_buckets; h_pos+=blockDim.x) // Clear local histogram
    SHist[h_pos] = 0;

  if (blockIdx.x*blockDim.x+threadIdx.x < chunk_a_size) { // If this thread has an atom
    int chunk_b_blocks = chunk_b_size/blockDim.x + 1;
    for(i=0; i < chunk_b_blocks-1; i++) // Loop through all but last block in chunk b
      {
        block_to_block(my_block,
                       &chunk_a[i*blockDim.x],
                       blockDim.x,
                       SHist,
                       histogram_resolution); // Compare my block to this block
      }
    block_to_block(my_block,
                   &chunk_a[i*blockDim.x],
                   chunk_b_size-i*blockDim.x,
                   SHist,
                   histogram_resolution); // Handle last block in chunk b (which may be small)
  }
  __syncthreads();
  for(int h_pos = threadIdx.x; h_pos < num_buckets; h_pos += blockDim.x)
    *(histogram_GPU+(num_buckets*blockIdx.x)+h_pos) += SHist[h_pos]; // Commit local histogram to private histogram
}

__global__ void GPUIntraChunkKernel (unsigned long long chunk_size, float histogram_resolution, atom * chunk_a, unsigned long long * histogram_GPU, int num_buckets) {

  extern __shared__ unsigned long long SHist[];
	int i, h_pos;
	float dist;
  atom * my_block = &chunk_a[blockIdx.x * blockDim.x]; // Pointer to this blocks atoms in chunk a
  atom temp_atom_1 = my_block[threadIdx.x]; // This thread's atom

  for(h_pos=threadIdx.x; h_pos < num_buckets; h_pos+=blockDim.x) // Clear local histogram
    SHist[h_pos] = 0;

  __syncthreads();

	/* loop through all points in atom list calculating distance from current point to all further points in this block*/
  for (i = threadIdx.x + 1; i < blockDim.x && i+blockIdx.x*blockDim.x < chunk_size; i++)
  {
    atom temp_atom_2 = my_block[i];
    dist = sqrt((temp_atom_1.x_pos - temp_atom_2.x_pos) * (temp_atom_1.x_pos - temp_atom_2.x_pos) +
                (temp_atom_1.y_pos - temp_atom_2.y_pos) * (temp_atom_1.y_pos - temp_atom_2.y_pos) +
                (temp_atom_1.z_pos - temp_atom_2.z_pos) * (temp_atom_1.z_pos - temp_atom_2.z_pos));
    h_pos = (int)(dist / histogram_resolution);
    atomicAdd(&(SHist[h_pos]), 1);
  }
  __syncthreads();
  /* Loop through all but last remaining blocks and compare this block with it*/
  for(i=blockIdx.x+1; i < gridDim.x-1; i++)
    block_to_block(my_block,
                   &chunk_a[i*blockDim.x],
                   blockDim.x,
                   SHist,
                   histogram_resolution);
  block_to_block(my_block,
                 &chunk_a[i*blockDim.x],
                 chunk_size-i*blockDim.x, // Last block may be small
                 SHist,
                 histogram_resolution);
  __syncthreads();
  for(h_pos = threadIdx.x; h_pos < num_buckets; h_pos += blockDim.x)
    *(histogram_GPU+(num_buckets*blockIdx.x)+h_pos) += SHist[h_pos]; // Commit local histogram to private histogram
}

/* print the counts in all buckets of the histogram  */
void output_histogram_GPU(){
	int i;
	unsigned long long total_cnt = 0;
	for(i=0; i< num_buckets; i++) {
		if(i%5 == 0) /* we print 5 buckets in a row */
			printf("\n%02d: ", i);
		printf("%15lld ", histogram[i]);
		total_cnt += histogram[i];
	  	/* we also want to make sure the total distance count is correct */
		if(i == num_buckets - 1)
			printf("\n T:%lld \n", total_cnt);
		else printf("| ");
	}
}

void GPU_baseline() {

  int num_chunks = ((PDH_acnt + CHUNK_SIZE)/CHUNK_SIZE);
  int num_blocks = ((PDH_acnt + block_size)/block_size);
  

	/* copy histogram to device memory */
	hipMalloc((void**) &histogram_GPU, sizeof(unsigned long long)*num_buckets);
	hipMemset(histogram_GPU, 0, sizeof(unsigned long long)*num_buckets);
	hipMalloc((void**) &temp_interchunk_histogram_GPU, sizeof(unsigned long long)*num_buckets*num_blocks);
	hipMemset(temp_interchunk_histogram_GPU, 0, sizeof(unsigned long long)*num_buckets*num_blocks);
	hipMalloc((void**) &temp_intrachunk_histogram_GPU, sizeof(unsigned long long)*num_buckets*num_blocks);
	hipMemset(temp_intrachunk_histogram_GPU, 0, sizeof(unsigned long long)*num_buckets*num_blocks);
	
	/* start time keeping */
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord( start, 0 );

	/* copy atom list to device memory */
	hipMalloc((void**) &chunk_a, sizeof(atom) * CHUNK_SIZE);
	hipMalloc((void**) &chunk_b, sizeof(atom) * CHUNK_SIZE);

	/* Run Kernel */
	for(int i=0;i<num_chunks;i++){ // Loop over all chunks
    int size_a = (i==num_chunks-1) ? PDH_acnt-i*CHUNK_SIZE : CHUNK_SIZE; // Last chunk may be small
    hipMemcpy(chunk_a, &atom_list[i*CHUNK_SIZE], sizeof(atom) * size_a, hipMemcpyHostToDevice); // Copy to chunk a
    // Handle comparisons internal to this chunk
    GPUIntraChunkKernel<<<num_blocks, block_size, sizeof(unsigned long long)*num_buckets>>>(size_a, PDH_res, chunk_a, temp_intrachunk_histogram_GPU, num_buckets);
    for(int j=i+1; j<num_chunks;j++){ // Loop through remaining chunks
      int size_b = (j==num_chunks-1) ? PDH_acnt-j*CHUNK_SIZE : CHUNK_SIZE; // Last chunk may be small
      hipMemcpy(chunk_b, &atom_list[j*CHUNK_SIZE], sizeof(atom) * size_b, hipMemcpyHostToDevice); // Copy to chunk b
      // Compare chunk a to chunk b
      GPUInterChunkKernel<<<num_blocks, block_size, sizeof(unsigned long long)*num_buckets>>>(size_a, size_b, PDH_res, chunk_a, chunk_b, temp_interchunk_histogram_GPU, num_buckets);
    }
  }

  hipDeviceSynchronize();
  kernelSumHistogram<<<3, 512>>>(temp_interchunk_histogram_GPU, histogram_GPU, PDH_acnt, num_buckets, block_size); // Add internal comparisons to histogram
  hipDeviceSynchronize();
  kernelSumHistogram<<<3, 512>>>(temp_intrachunk_histogram_GPU, histogram_GPU, PDH_acnt, num_buckets, block_size); // Add chunk-to-chunk comparisons to histogram

	/* stop time keeping */
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	float elapsedTime;
	hipEventElapsedTime( &elapsedTime, start, stop );

	/* transfer histogram to host memory */
	hipMemcpy(histogram, histogram_GPU, sizeof(unsigned long long)*num_buckets, hipMemcpyDeviceToHost);
	
	/* print out the histogram */
	output_histogram_GPU();
	elapsedTime = elapsedTime/1000;
	printf( "******** Total Running Time of Kernel = %0.5f sec *******\n", elapsedTime );

	/* free cuda timekeeping */
	hipEventDestroy( start );
	hipEventDestroy( stop );
	
  hipFree(temp_intrachunk_histogram_GPU);
  hipFree(temp_interchunk_histogram_GPU);
  hipFree(chunk_a);
  hipFree(chunk_b);
  hipFree(histogram_GPU);
}

/* Input Validation Function */
bool isNumber(char number[], bool floatingpoint)
{
    for (int i = 0; number[i] != 0; i++)
    {
        //if (number[i] > '9' || number[i] < '0')
        if (!isdigit(number[i]))
		{
			if((number[i] == '.' && floatingpoint))
			{
				floatingpoint = false;
			}
			else
			{
				return false;
			}
		}
    }
    return true;
}


/* Most of this input validation can probably be pulled whenever we hardcode our block size and if we hardcode our bucket width */
int main(int argc, char **argv)
{
	/* input validation */
	if((argc > 3))
	{
		if(((isNumber(argv[1], false) && isNumber(argv[2], true)) && isNumber(argv[3], false)))
		{
			PDH_acnt = atoi(argv[1]);
			PDH_res	 = atof(argv[2]);
			block_size = atoi(argv[3]);
		}
		else
		{
			printf( "Invalid Input Error Invalid Arguments\n Valid input is ./program_name {#of_samples} {bucket_width} {block_size}\n");
			return 0;
		}
	}
	else
	{
		printf( "Invalid Input Error Insufficient Arguments\n Valid input is ./program_name {#of_samples} {bucket_width} {block_size}\n");
		return 0;
	}

	/* allocate memory */
	num_buckets = (int)(BOX_SIZE * 1.732 / PDH_res) + 1;
	histogram = (unsigned long long *)malloc(sizeof(unsigned long long)*num_buckets);
	atom_list = (atom *)malloc(sizeof(atom)*PDH_acnt);

	srand(1);
	/* generate data following a uniform distribution */
	for(int i = 0;  i < PDH_acnt; i++) {
		atom_list[i].x_pos = ((float)(rand()) / RAND_MAX) * BOX_SIZE;
		atom_list[i].y_pos = ((float)(rand()) / RAND_MAX) * BOX_SIZE;
		atom_list[i].z_pos = ((float)(rand()) / RAND_MAX) * BOX_SIZE;
	}

	/* call GPU histrogram compute */
	GPU_baseline();

	/* free memory */
	free(histogram);
	free(atom_list);

	return 0;
}


